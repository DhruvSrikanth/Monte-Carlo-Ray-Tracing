#include <iostream>
using namespace std;
#include <assert.h>
#include <math.h>
#include <fstream>
#include <random>
#include<hip/hip_runtime.h>


// Maximum number of blocks in a device grid (for dim x)
#define MAX_BLOCKS 2147483647

// Min function
#define MIN(a,b) (((a)<(b))?(a):(b))

// Data structure for a point
struct Point {
    // x, y, z coordinates
    double x;
    double y;
    double z;
};

// HOST FUNCTIONS

void print_point(Point p) {
    // Print the point
    cout << "(" << p.x << ", " << p.y << ", " << p.z << ")" << endl;
}

void write_to_file(double* output, string filename, int N, int NT) { 
    // Allocate memory for the file
    ofstream file;
    file.open(filename);

    // Write the timestep to the file
    file << "[" << NT << "], ";

    // Write the data to the file
    file << "[";
    int n = sqrt(N);
    for (int i = 0; i < n; i++) {
        if (i == 0) {
            file << "[";
        } 
        else {
            file << ", [";
        }
        for (int j = 0; j < n; j++) {
            int idx_1d = i * n + j;
            if (j == N - 1) {
                file << output[idx_1d];
            } 
            else {
                file << output[idx_1d] << ", ";
            }
        }
        file << "]";
    }
    file << "]";

    // Release the memory for the file
    file.close();
}

void create_contiguous_2d_array(double* mat, int N) {
    // Create a contiguous 2d array
    int n = sqrt(N);
    // Initialize the matrix
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            int idx_1d = i*n +j;
            mat[idx_1d] = 0.0;
        }
    }
}

// DEVICE FUNCTIONS

__device__ Point vec_add(Point p1, Point p2) {
    // Add two vectors
    Point res;
    res.x = p1.x + p2.x;
    res.y = p1.y + p2.y;
    res.z = p1.z + p2.z;
    return res;
}

__device__ Point vec_scale(Point p, double scale) {
    // Scale a vector
    Point res;
    res.x = p.x * scale;
    res.y = p.y * scale;
    res.z = p.z * scale;
    return res;
}

__device__ double vec_dotp(Point p1, Point p2) {
    // Dot product of two vectors
    double res;
    res = p1.x * p2.x + p1.y * p2.y + p1.z * p2.z;
    return res;
}

__device__ Point vec_direction(Point p1, Point p2) {
    // Direction of p2 from p1
    Point res;
    double magnitude = sqrt(pow(p2.x - p1.x, 2) + pow(p2.y - p1.y, 2) + pow(p2.z - p1.z, 2));
    res.x = (p2.x - p1.x) / magnitude;
    res.y = (p2.y - p1.y) / magnitude;
    res.z = (p2.z - p1.z) / magnitude;
    return res;
}

__device__ double LCG_random_double(uint64_t& seed) {
    // Generate a random number between 0 and 1 from a uniform distribution
    const uint64_t m = 9223372036854775808ULL; // 2ˆ63
    const uint64_t a = 2806196910506780709ULL;
    const uint64_t c = 1ULL;
    seed = (a * seed + c ) % m;
    return (double) (seed) / (double) m;
}

__device__ uint64_t fast_forward_LCG(uint64_t seed, uint64_t n) {
    // Fast forward the LCG by n steps
    const uint64_t m = 9223372036854775808ULL; // 2ˆ63
    uint64_t a = 2806196910506780709ULL;
    uint64_t c = 1ULL;
    n = n % m;
    uint64_t a_new = 1;
    uint64_t c_new = 0;
    while (n >0) {
        if (n & 1) {
            a_new *= a;
            c_new = c_new * a + c;
        }
        c *= ( a + 1);
        a *= a;
        n >>= 1;
    }
    return (a_new * seed + c_new) % m;
}

__device__ Point direction_sampling(uint64_t& seed) {
    // Generate a random ray
    Point V;

    // Sample point for respective distributions
    double phi = 2*M_PI*LCG_random_double(seed); // 0 ~ 2*pi
    double cos_theta = 2*LCG_random_double(seed) - 1; // -1 ~ 1
    double sin_theta = sqrt(1 - pow(cos_theta, 2));
    
    V.x = sin_theta * cos(phi);
    V.y = sin_theta * sin(phi);
    V.z = cos_theta;

    return V;
}

__global__ void ray_tracing(double* grid, int* N_gridpoints) {
    // Initialize points
    Point W, V, I, N, S;

    // Initialize simulation parameters
    double w_max = 10.0;
    Point L = {4,4,-1};
    Point C = {0,12,0};
    double r = 6.0;
    double Wy = 10.0;

    // Initialize variables
    double t;
    double b;

    uint64_t init_seed = 1;
    uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t n = 200 * tid;

    while (true) {
        // Generate random seed
        uint64_t seed = fast_forward_LCG(init_seed, n);
        
        // sample random v from unit sphere
        V = direction_sampling(seed);
        W = vec_scale(V, Wy / V.y);
        bool condition = abs(W.x) < w_max && abs(W.z) < w_max && pow(vec_dotp(V, C), 2) + r*r - vec_dotp(C, C) > 0;
        if (condition) {
            break;
        }
        init_seed = seed;
    }

    t = vec_dotp(V,C) - sqrt(pow(vec_dotp(V,C), 2) + r*r - vec_dotp(C, C));
    I = vec_scale(V, t);
    N = vec_direction(C, I);
    S = vec_direction(I, L);
    b = max(0.0, vec_dotp(S, N));

    // Compute the grid point indices
    int i = (W.z + w_max) / (2*w_max) * (*N_gridpoints);
    int j = (W.x + w_max) / (2*w_max) * (*N_gridpoints);
    int idx_1d_local = i * (*N_gridpoints) + j;

    // Update the grid point
    atomicAdd(&grid[idx_1d_local], b);
    // grid[idx_1d_local] += b; // For the serial version
}

int main(int argc, char** argv) {
    // Initialize variables
    int N_rays = stod(argv[1]);
    int N_gridpoints = stoi(argv[2]);
    int n_threads_per_block = stoi(argv[3]);

    // Compute the number of blocks
    int n_blocks = MIN(N_rays/n_threads_per_block + 1, MAX_BLOCKS);

    cout << "Simulation Parameters:" << endl;
    cout << "Number of rays = " << N_rays << endl;
    cout << "Number of gridpoints = " << N_gridpoints << endl;

    // Simple checks
    assert(N_rays > 0);
    assert(N_gridpoints > 0);

    // Initialize grid
    double* grid = new double[N_gridpoints*N_gridpoints];
    create_contiguous_2d_array(grid, N_gridpoints*N_gridpoints);

    // Allocate memory for the number of grid points and the grid
    double* grid_device;
    int* N_gridpoints_device;

    hipMalloc(&grid_device, N_gridpoints*N_gridpoints*sizeof(double));
    hipMalloc(&N_gridpoints_device, sizeof(int));

    // Copy the params to the device
    hipMemcpy(grid_device, grid, N_gridpoints*N_gridpoints*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(N_gridpoints_device, &N_gridpoints, sizeof(int), hipMemcpyHostToDevice);

    // CUDA timer
    hipEvent_t start_device, stop_device;  
    float time_device;

    // Create timers
    hipEventCreate(&start_device);
    hipEventCreate(&stop_device);

    // Start timer
    hipEventRecord(start_device, 0);  
    
    // Perform simulation
    ray_tracing<<<n_blocks, n_threads_per_block>>>(grid_device, N_gridpoints_device);

    // Stop timer
    hipEventRecord(stop_device, 0);
    hipEventSynchronize(stop_device);
    hipEventElapsedTime(&time_device, start_device, stop_device);

    // Print out metrics
    cout << "Grind Rate: " << N_rays / floor(1e3/time_device) << " rays/sec" << endl; 
    cout << "Time: " << time_device << " ms" << endl;
    
    // Copy the grid from the device to the host
    hipMemcpy(grid, grid_device, N_gridpoints*N_gridpoints*sizeof(double), hipMemcpyDeviceToHost);

    // Release the memory for the timer
    hipEventDestroy(start_device);
    hipEventDestroy(stop_device);

    // Write to file
    write_to_file(grid, "./output/output.txt", N_gridpoints*N_gridpoints, N_rays-1);

    // Release the memory for the grid and the number of grid points
    hipFree(grid_device);
    hipFree(N_gridpoints_device);
    delete[] grid;

    return 0;

}